#if 1
#include "hip/hip_runtime.h"
#include ""
#include "cuda_localmatch.cuh"
#include <stdio.h>
#include <opencv2/opencv.hpp>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

int main()
{
	// host memory
	cv::Mat left = cv::imread("left.png", 0);
	cv::Mat right = cv::imread("right.png", 0);
	assert(left.cols > 0 && right.cols > 0);
	assert(left.size() == right.size());

	int height = left.rows;
	int width = left.cols;

	// device memory
	uchar *d_left, *d_right, *d_out;
	hipMalloc((void**)&d_left, width*height);
	hipMalloc((void**)&d_right, width*height);
	hipMalloc((void**)&d_out, width*height);

	hipMemcpy(d_left, left.data, width*height, hipMemcpyHostToDevice);
	hipMemcpy(d_right, right.data, width*height, hipMemcpyHostToDevice);
	hipMemset(d_out, 0, width*height * sizeof(uchar));

	// launch kernel
	dim3 block_size, grid_size;
	block_size = dim3(32, 32, 1);
	grid_size = dim3((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y, 1);

	SADMatch << <grid_size, block_size >> > (d_left, d_right, d_out, 64, 5, width, height);

	// copy result back
	cv::Mat result_disparity(height, width, CV_8UC1);
	hipMemcpy(result_disparity.data, d_out, width*height * sizeof(uchar), hipMemcpyDeviceToHost);

	hipFree(d_left);
	hipFree(d_right);
	hipFree(d_out);
	cv::imshow("result", result_disparity);

	cv::waitKey(0);
}


int main_t(int argc, char *argv[])
{
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	char result[100];
	sprintf(result, "{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	printf(result);
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
#endif